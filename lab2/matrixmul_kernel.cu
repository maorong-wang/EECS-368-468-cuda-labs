#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	//use padding here to avoid bank conflict
	__shared__ float Ms[17*16];
	__shared__ float Ns[17*16];
	__shared__ double Ps[17*16];
	unsigned int xIndex = __mul24(blockDim.x,blockIdx.x)+threadIdx.x;
	unsigned int yIndex = __mul24(blockDim.y,blockIdx.y)+threadIdx.y;
	
	int iter;
	Ps[(blockDim.x+1)*threadIdx.y+threadIdx.x]=0.0f;
	
	for(iter=0;iter<1+(M.width-1)/16;iter++)
	{
		if(xIndex<N.width || yIndex < M.height)
		{
			Ms[threadIdx.y*(blockDim.x+1)+threadIdx.x] = M.elements[yIndex*M.width+iter*blockDim.x+threadIdx.x];
			Ns[threadIdx.y*(blockDim.x+1)+threadIdx.x] = N.elements[xIndex+(iter*blockDim.y+threadIdx.y)*N.width];
		}
		else
		{
			Ms[threadIdx.y*(blockDim.x+1)+threadIdx.x] = 0.0f;
			Ns[threadIdx.y*(blockDim.x+1)+threadIdx.x] = 0.0f;	
		}
		__syncthreads();
		
		if(xIndex<N.width && yIndex < M.height)
		{
			int temp;
			for(temp=0;temp<blockDim.x;temp++)
			{
				Ps[(blockDim.x+1)*threadIdx.y+threadIdx.x] += Ms[threadIdx.y*(blockDim.x+1)+temp] * Ns[temp*(blockDim.x+1)+threadIdx.x];
			}
		}
		__syncthreads();
	}


	if(xIndex<N.width && yIndex < M.height)
	{
		P.elements[yIndex*P.width+xIndex]=Ps[threadIdx.y*(blockDim.x+1)+threadIdx.x];
	}
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
